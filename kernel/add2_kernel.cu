
#include <hip/hip_runtime.h>
//实现两个长度为n的tensor的相加, add2_kernel是kernel函数，运行在GPU端的
__global__ void add2_kernel(float* c,
                            const float* a,
                            const float* b,
                            int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
            i < n; i += gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}

//每个block有1024个线程，一共有n/1024个block。launch_add2是CPU端的执行函数，调用kernel。
void launch_add2(float* c,
                 const float* a,
                 const float* b,
                 int n) {
    dim3 grid((n + 1023) / 1024);
    dim3 block(1024);
    add2_kernel<<<grid, block>>>(c, a, b, n);
}